#include "hip/hip_runtime.h"

#include <stdio.h>

__global__
void batchStridedSum(
        float *input, float *output,
        int batchSize, int rows, int cols) {
    for (int i = 0; i < batchSize; i++) {
        for (int j = 0; j < (rows * cols); j++) {
            output[j] = output[j] + input[i * rows * cols + j];
        }
    }
}


extern "C" {
size_t cuBatchStridedSum(
        float *inputPtr, float *outputPtr,
        int batchSize, int rows, int cols
) {

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    batchStridedSum<<<1, 1>>>(inputPtr, outputPtr, batchSize, rows, cols);

    if (cudaStatus != hipSuccess) {
        goto Error;
    }
    hipDeviceSynchronize();
    return 0;

    Error:
    return 1;
}
} // extern C
