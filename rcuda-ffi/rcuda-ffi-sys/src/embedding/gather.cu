#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__
void CuSubset(
        float *embedding, float *words, float *embeddingOutput,
        int phraseLength,
        int embedDimension,
        int vocabSize,
        int batchSize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < (phraseLength * batchSize); i += stride) {
        // TODO: Add in boolean mask to remove branching
        for (int j = 0; j < embedDimension; j++) {
            int embedIndex = (j * vocabSize) + (int) words[i] - 1;
            int outputIndex = embedDimension * i + j;
            if (((int) words[i] != 0) && (embedIndex < vocabSize * embedDimension)) {
                embeddingOutput[outputIndex] = embedding[embedIndex];
            } else {
                embeddingOutput[outputIndex] = 0.0;
            }
        }
    }
}


extern "C" {
size_t cuGather(
        int embedDimension, int phraseLength,
        int vocabSize, int batchSize,
        float *src_ptr,
        float *weight_ptr,
        float *dest_ptr
) {

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    int blockSize = 256;
    int numBlocks = (phraseLength + blockSize - 1) / blockSize;
    CuSubset<<<numBlocks, blockSize>>>(
            weight_ptr,
            src_ptr,
            dest_ptr,
            phraseLength,
            embedDimension,
            vocabSize,
            batchSize);

    if (cudaStatus != hipSuccess) {
        goto Error;
    }
    hipDeviceSynchronize();
    return 0;

    Error:
    return 1;
}
} // extern C
